#include "hip/hip_runtime.h"
﻿#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime_api.h>
#include <>
#include <hip/driver_types.h>
#include "hip/hip_runtime.h"
#include "vector_add.h"

//Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError(R) {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   return R; \
 }                                                                 \
}

/// <summary>
/// Vector Add Kernel that executes on device
/// </summary>
__global__ void vector_add_kernel(const float *a, const float *b, float  *c, unsigned int N)
{
	// Index calculation
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	// Safe-check for any extra launched threads
	if(idx < N)
	{
		c[idx] = a[idx] + b[idx];
	}
}

/// <summary>
/// Launches the test case
/// </summary>
int vector_add::runner()
{
	system("CLS");
	printf("\n\n\n[VECTOR_ADD]: STARTING vector_add example.\n");
	
	// -------------------------------------
	// Definitions
	printf("Initizialing definitions\n");

	const unsigned int no_of_elements = 128000000; // amount of total elements in vectors
	const size_t size = no_of_elements * sizeof(float); // required size
	const int threads_per_block = 512; // threads per block
	const int blocks = (int)ceil((float)no_of_elements / threads_per_block); // calculate required blocks

	float ms = 0, total_ms = 0;
	hipEvent_t kernel_start, kernel_end, memcpy_to_start, memcpy_to_end, memcpy_from_start, memcpy_from_end; // Timing variables
	hipEventCreate(&kernel_start);
	hipEventCreate(&kernel_end);
	hipEventCreate(&memcpy_to_start);
	hipEventCreate(&memcpy_to_end);
	hipEventCreate(&memcpy_from_start);
	hipEventCreate(&memcpy_from_end);

	printf("[Settings] Elements#: %d | reqired size: %d bytes | threads per block: %d | calculated blocks: %d\n\n", no_of_elements, size, threads_per_block, blocks);
	// -------------------------------------


	// -------------------------------------
	// Declare and allocate memory on HOST
	printf("Allocating memory on HOST\n");
	float* h_a = static_cast<float*>(malloc(size));
	float* h_b = static_cast<float*>(malloc(size));
	float* h_c = static_cast<float*>(malloc(size));
	if (h_a == nullptr || h_b == nullptr || h_c == nullptr) return -1;
	printf("DONE\n\n");
	// -------------------------------------


	// -------------------------------------
	// Declare and allocate memory on DEVICE
	printf("Allocating memory on DEVICE\n");
	float* d_a, * d_b, * d_c;
	hipMalloc(reinterpret_cast<void**>(&d_a), size);
	cudaCheckError(-2);
	hipMalloc(reinterpret_cast<void**>(&d_b), size);
	cudaCheckError(-2);
	hipMalloc(reinterpret_cast<void**>(&d_c), size);
	cudaCheckError(-2);
	printf("DONE\n\n");
	// -------------------------------------


	// -------------------------------------
	// Init HOST input vector data
	printf("Initializing HOST input vectors (all set to 1.0f)\n");
	for (int i = 0; i < no_of_elements; i++)
	{
		h_a[i] = 1.0f;
		h_b[i] = 1.0f;
	}
	printf("DONE\n\n");
	// -------------------------------------


	// -------------------------------------
	// Copy HOST Input vectors to device
	printf("COPYING input data from HOST to DEVICE\n");
	hipEventRecord(memcpy_to_start);
	cudaCheckError(-3);
	hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	cudaCheckError(-3);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	cudaCheckError(-3);
	hipEventRecord(memcpy_to_end);
	cudaCheckError(-3);
	printf("DONE\n\n");
	// -------------------------------------


	// -------------------------------------
	// Kernel Launch
	printf("LAUNCHING Kernel\n");
	hipEventRecord(kernel_start);
	cudaCheckError(-4);
	vector_add_kernel << <blocks, threads_per_block >> > (d_a, d_b, d_c, no_of_elements);
	cudaCheckError(-4);
	printf("WAITING for kernel to finish execution\n");
	hipDeviceSynchronize(); // BARRIER - Wait for kernel to finish execution
	cudaCheckError(-4);
	hipEventRecord(kernel_end);
	cudaCheckError(-4);
	printf("KERNEL finished executing\n");
	// -------------------------------------


	// -------------------------------------
	// Copy results back to HOST
	printf("COPYING result data from DEVICE to HOST\n");
	hipEventRecord(memcpy_from_start);
	cudaCheckError(-5);
	hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
	cudaCheckError(-5);
	hipEventRecord(memcpy_from_end);
	cudaCheckError(-5);
	printf("DONE\n\n");
	// -------------------------------------

	// -------------------------------------
	// Print result
	printf("Device results sample:\n");
	for (int i = 0; i < 10; i++)
	{
		printf("%0.2f ", h_c[i]);
	}
	printf("...\n\n");
	// -------------------------------------


	// -------------------------------------
	// Time events
	hipEventElapsedTime(&ms, memcpy_to_start, memcpy_to_end);
	cudaCheckError(-6);
	total_ms += ms;
	printf("Memcpy from HOST to DEVICE time: %f sec\n", ms / 1000.0);

	hipEventElapsedTime(&ms, kernel_start, kernel_end);
	cudaCheckError(-6);
	total_ms += ms;
	printf("KERNEL execution time: %f sec\n", ms / 1000.0);
	
	hipEventElapsedTime(&ms, memcpy_from_start, memcpy_from_end);
	cudaCheckError(-6);
	total_ms += ms;
	printf("Memcpy from DEVICE to HOST time: %f sec\n\n", ms / 1000.0);
	
	printf("TOTAL Execution time: %f sec\n", total_ms / 1000.0);
	// -------------------------------------

	// -------------------------------------
	// Free memory
	hipFree(d_a);
	cudaCheckError(-7);
	hipFree(d_b);
	cudaCheckError(-7);
	hipFree(d_c);
	cudaCheckError(-7);

	free(h_a);
	free(h_b);
	free(h_c);
	// -------------------------------------

	return 0;
}
